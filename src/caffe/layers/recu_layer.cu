#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
__global__ void ReCUForward(const int n, const Dtype* bottom, Dtype* top,
    Dtype negative_slope, Dtype xc, Dtype yc, Dtype s, Dtype ys, Dtype beta);

template <>
__global__ void ReCUForward<float>(const int n, const float* bottom, float* top,
    float negative_slope, float xc, float yc, float s, float ys, float beta) {
  CUDA_KERNEL_LOOP(index, n) {
    if (bottom[index] <= 0) {
      top[index] = negative_slope * bottom[index];
    }
    else if (bottom[index] <= s) {
      top[index] = sqrtf(xc * xc + yc * yc - (bottom[index] - xc) * (bottom[index] - xc)) + yc;
    }
    else {
      top[index] = ys + (bottom[index] - s) * beta;
    }   
  }
}
template <>
__global__ void ReCUForward<double>(const int n, const double* bottom, double* top,
    double negative_slope, double xc, double yc, double s, double ys, double beta) {
  CUDA_KERNEL_LOOP(index, n) {
    if (bottom[index] <= 0) {
      top[index] = negative_slope * bottom[index];
    }
    else if (bottom[index] <= s) {
      top[index] = sqrt(xc * xc + yc * yc - (bottom[index] - xc) * (bottom[index] - xc)) + yc;
    }
    else {
      top[index] = ys + (bottom[index] - s) * beta;
    }   
  }
}
template <typename Dtype>
void ReCULayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    vector<Blob<Dtype>*>* top) {
  if (Caffe::phase() == Caffe::TEST) {
    // this->Analysis(bottom, top);
  }
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = (*top)[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  Dtype negative_slope = this->layer_param_.recu_param().negative_slope();
  // NOLINT_NEXT_LINE(whitespace/operators)
  ReCUForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, top_data, negative_slope, xc, yc, s, ys, beta);
  CUDA_POST_KERNEL_CHECK;
  // << " count: " << count << " bottom_data: "
  //     << (unsigned long)bottom_data
  //     << " top_data: " << (unsigned long)top_data
  //     << " blocks: " << CAFFE_GET_BLOCKS(count)
  //     << " threads: " << CAFFE_CUDA_NUM_THREADS;
}

template <typename Dtype>
__global__ void ReCUBackward(const int n, const Dtype* top_diff,
    const Dtype* bottom_data, Dtype* bottom_diff, Dtype negative_slope, Dtype xc, Dtype yc, Dtype s, Dtype ys, Dtype beta);

template <>
__global__ void ReCUBackward<float>(const int n, const float* top_diff,
    const float* bottom_data, float* bottom_diff, float negative_slope, float xc, float yc, float s, float ys, float beta) {
  CUDA_KERNEL_LOOP(index, n) {
    if (bottom_data[index] <= 0) {
      bottom_diff[index] = negative_slope * top_diff[index];
    }
    else if (bottom_data[index] <= s) {
      bottom_diff[index] = top_diff[index] * (xc - bottom_diff[index]) / sqrtf(-bottom_diff[index] * bottom_diff[index] + 2 * xc * bottom_diff[index] + yc * yc);
    }
    else {
      bottom_diff[index] = top_diff[index] * beta;
    }   
  }
}

template <>
__global__ void ReCUBackward<double>(const int n, const double* top_diff,
    const double* bottom_data, double* bottom_diff, double negative_slope, double xc, double yc, double s, double ys, double beta) {
  CUDA_KERNEL_LOOP(index, n) {
    if (bottom_data[index] <= 0) {
      bottom_diff[index] = negative_slope * top_diff[index];
    }
    else if (bottom_data[index] <= s) {
      bottom_diff[index] = top_diff[index] * (xc - bottom_diff[index]) / sqrt(-bottom_diff[index] * bottom_diff[index] + 2 * xc * bottom_diff[index] + yc * yc);
    }
    else {
      bottom_diff[index] = top_diff[index] * beta;
    }   
  }
}

template <typename Dtype>
void ReCULayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    vector<Blob<Dtype>*>* bottom) {
  if (propagate_down[0]) {
    const Dtype* bottom_data = (*bottom)[0]->gpu_data();
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = (*bottom)[0]->mutable_gpu_diff();
    const int count = (*bottom)[0]->count();
    Dtype negative_slope = this->layer_param_.recu_param().negative_slope();
    // NOLINT_NEXT_LINE(whitespace/operators)
    ReCUBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, bottom_data, bottom_diff, negative_slope, xc, yc, s, ys, beta);
    CUDA_POST_KERNEL_CHECK;
  }
}


INSTANTIATE_CLASS(ReCULayer);


}  // namespace caffe
